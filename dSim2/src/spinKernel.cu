#include "hip/hip_runtime.h"
/* 
 * MR Diffusion spin system code
 * ----------------------------
 *
 * This spin system class is implemented here.
 *
 * Copyright 2008 Bob Dougherty (bobd@stanford.edu) and 
 * Shyamsundar Gopalakrishnan (gshyam@stanford.edu).
 */
 
#ifndef _SPIN_KERNEL_H_
#define _SPIN_KERNEL_H_

#include <stdio.h>
#include <math.h>
#include <hip/hip_vector_types.h>
#include "hip/hip_math_constants.h"
#include "spinKernel.cuh"
#include "options.h"

#define PI 3.14159265358979f
#define TWOPI 6.28318530717959f

texture<float4,1,hipReadModeElementType> texFiberList;
texture<uint, 1,hipReadModeElementType> texCubeList;

typedef unsigned int uint;

// Some simple vector ops for float3's (dot and length are defined in cudautil_math)
//#define dot(u,v)   ((u).x * (v).x + (u).y * (v).y + (u).z * (v).z)
//#define length(v)    sqrt(dot(v,v))  // norm (vector length)
#define d(u,v)     length(u-v)       // distance (norm of difference)

// Returns the shortest distance from a point P to a line defined
// by two points (LP1 and LP2)
__device__ float point_line_dist(float3 P, float3 LP1, float3 LP2){ 
   float3 v = LP2-LP1; 
   float b = dot(P-LP1, v) / dot(v, v);
   return d(P, LP1+b*v);
}

// Returns the shortest distance from a point P to a line segment
// defined by two points (SP1 and SP2)
__device__ float point_seg_dist(float3 P, float3 SP1, float3 SP2){
    float3 v = SP2-SP1;
    float c1 = dot(P-SP1, v);
    if(c1<=0) return d(P, SP1);
    float c2 = dot(v, v);
    if(c2<=c1) return d(P, SP2);
    float3 Pb = SP1 + c1/c2 * v;
    return d(P, Pb);
}

__device__ void boxMuller(float& u1, float& u2){ // num of ops = 5
    float   r = sqrtf(-2.0f * __logf(u1));
    float phi = TWOPI * u2;
    u1 = r * __cosf(phi);
    u2 = r * __sinf(phi);
}

/*__device__ uint rand31pmc(uint &seed){//num of ops = 5
   uint hi, lo;
   lo = 16807 * (seed & 0xFFFF);
   hi = 16807 * (seed >> 16);
   lo += (hi & 0x7FFF) << 16;
   lo += hi >> 15;                  
   if (lo > 0x7FFFFFFF) lo -= 0x7FFFFFFF;          
   return ( seed = lo );        
}*/

/*
 * A faster 48-bit PNRG from Arnold and van Meel (released under GPL). 
  Copyright (c) 2007 A. Arnold and J. A. van Meel, FOM institute
  AMOLF, Amsterdam; all rights reserved unless otherwise stated.
  "Harvesting graphics power for MD simulations"
  by J.A. van Meel, A. Arnold, D. Frenkel, S. F. Portegies Zwart and
  R. G. Belleman, arXiv:0709.3225.
 
 * propagate an rand48 RNG one iteration.
    @param Xn  the current RNG state, in 2x 24-bit form
    @param A,C the magic constants for the RNG. For striding,
               this constants have to be adapted, see the constructor
    @result    the new RNG state X(n+1)
*/
/*__device__
static uint2 RNG_rand48_iterate_single(uint2 Xn, uint2 A, uint2 C){
  // results and Xn are 2x 24bit to handle overflows optimally, i.e.
  // in one operation.

  // the multiplication commands however give the low and hi 32 bit,
  // which have to be converted as follows:
  // 48bit in bytes = ABCD EF (space marks 32bit boundary)
  // R0             = ABC
  // R1             =    D EF

  unsigned int R0, R1;

  // low 24-bit multiplication
  const unsigned int lo00 = __umul24(Xn.x, A.x);
  const unsigned int hi00 = __umulhi(Xn.x, A.x);

  // 24bit distribution of 32bit multiplication results
  R0 = (lo00 & 0xFFFFFF);
  R1 = (lo00 >> 24) | (hi00 << 8);

  R0 += C.x; R1 += C.y;

  // transfer overflows
  R1 += (R0 >> 24);
  R0 &= 0xFFFFFF;

  // cross-terms, low/hi 24-bit multiplication
  R1 += __umul24(Xn.y, A.x);
  R1 += __umul24(Xn.x, A.y);

  R1 &= 0xFFFFFF;

  return make_uint2(R0, R1);
}*/


__device__ uint myRand(uint seed[]){//num of ops = 5
   // Simple multiply-with-carry PRNG that uses two seeds (seed[0] and seed[1])
   // (Algorithm from George Marsaglia: http://en.wikipedia.org/wiki/George_Marsaglia)
    seed[0] = 36969 * (seed[0] & 65535) + (seed[0] >> 16);
    seed[1] = 18000 * (seed[1] & 65535) + (seed[1] >> 16);
    return (seed[0] << 16) + seed[1];
}

/* 
 * Return a random number r in the range 0<=r<=1
 */
__device__ float myRandf(uint seed[]){
    return((float)myRand(seed) / 4294967295.0f);
}

/* 
 * Return a vector with a specified magnitude (adc) and a random direction.
 */
__device__ void myRandDir(uint seed[], float adc, float3& vec){
    // azimuth and elevation are on the interval [0,2*pi)
    // (2*pi)/4294967294.0 = 1.4629181e-09f
    float az = (float)myRand(seed) * 1.4629181e-09f;
    float el = (float)myRand(seed) * 1.4629181e-09f;
    vec.z = adc * __sinf(el);
    float rcosel = adc * __cosf(el);
    vec.x = rcosel * __cosf(az);
    vec.y = rcosel * __sinf(az);
    return;
}

/*
 * returns three random numbers from the normal distribution (mean 0, std 1) and 
 * a forth from the uniform distribution.
 */
__device__ void myRandn(uint seed[], float& n1, float& n2, float& n3, float& u) {//num of ops = 8 + 4*5 + 5*2
  // We want random numbers in the range (0,1] (i.e., 0>n>=1):
  n1 = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
  n2 = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
  n3 = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
  u  = ((float)myRand(seed) + 1.0f) / 4294967296.0f;
  // Note that ULONG_MAX=4294967295
  float n4 = u;
  boxMuller(n1, n2);
  boxMuller(n3, n4);
  return;
}
	
// calculate position in uniform cube
__device__ int3 calcCubePos(float3 p,
                            float cubeLength
                            )
{//num of ops = 6
    int3 cubePos;
    cubePos.x = floor((p.x + 1.0f) / cubeLength);
    cubePos.y = floor((p.y + 1.0f) / cubeLength);
    cubePos.z = floor((p.z + 1.0f) / cubeLength);
    return cubePos;
}

// calculate address in cube from position (clamping to edges)

__device__ uint calcCubeHash(int3 cubePos,
                             uint numCubes)
{//num of ops = 8
    cubePos.x = max(0, min(cubePos.x, numCubes-1));
    cubePos.y = max(0, min(cubePos.y, numCubes-1));
    cubePos.z = max(0, min(cubePos.z, numCubes-1));
    return cubePos.z * numCubes * numCubes + cubePos.y* numCubes + cubePos.x;
}

// calculate position in uniform cube
__device__ int3 calcCubePos_4(float4 p,
                            float cubeLength
                            )
{//num of ops = 6
    int3 cubePos;
    cubePos.x = floor((p.x + 1.0f) / cubeLength);
    cubePos.y = floor((p.y + 1.0f) / cubeLength);
    cubePos.z = floor((p.z + 1.0f) / cubeLength);
    return cubePos;
}


/*
* Function: integrate()
* Return type: void
* Description: Computes the spin movement.
*
* oldPos is an array of float4's, where oldPos[i].x,y,z = 3d spatial coords of 
* spin i and oldPos[i].w is the phase of spin i.
*
* oldSeed is an array of uint4's where oldSeed[i].x,y are the two PRNG seeds 
* for spin i. oldSeed[i].
*
*/
__global__ void
integrate(  float4* oldPos, 
            uint4* oldSeed,
            float deltaTime,
            float permeability,
            float intraStdDev,
            float extraStdDev,
	    float myelinStdDev,          
            int numBodies,
            float gradX, float gradY, float gradZ,
            float4* fiberPos_1,
            uint* cubeCounters,
            uint* cubeList,
            float phaseConstant,
            float cubeLength,
            uint numCubes,
            uint maxFibersPerCube,
            float innerRadiusScale,
            uint iterations){
            
   int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

   if(index>=numBodies)
   return;

   float phase = oldPos[index].w;    
   float3 pos = make_float3(oldPos[index]);    
   //oldPos[index] = make_float4(pos.x+0.001f,pos.y-0.001f,pos.z+0.001f,phase);

   uint4 seed4 = oldSeed[index];
   uint insideFiberIndex = seed4.w;

   // Now apply the brownian motion (free diffusion). We simulate brownian motion
   // with a random walk where the x, y, and z componenets are drawn from a 
   // normal distribution with mean 0 and standard deviation of sqrt(2*ADC*deltaTime).
   // From wikipedia http://en.wikipedia.org/wiki/Random_walk:
   //    In 3D, the variance corresponding to the Green's function of the diffusion equation is:
   //       sigma^2 = 6*D*t
   //    sigma^2 corresponds to the distribution associated to the vector R that links the two 
   //    ends of the random walk, in 3D. The variance associated to each component Rx, Ry or Rz 
   //    is only one third of this value (still in 3D).
   // Thus, the standard deviation of each component is sqrt(2*ADC*deltaTime)

   uint rseed[2];
   rseed[0] = seed4.x;
   rseed[1] = seed4.y; // + clock() + (uint)index + 1000;
   
   bool isInside, reflected;
   
   for(uint i=0; i<iterations; i++){
      isInside = insideFiberIndex<UINT_MAX;
      // Take a random walk...
      // myRandn returns 3 PRNs from a normal distribution with mean 0 and SD of 1. 
      // So, we just need to scale these with the desired SD to get the displacements
      // for the random walk.
      // myRandn also returns a bonus uniformly distributed PRN as a side-effect of the 
      // Box-Muller transform used to generate normally distributed PRNs.
      float u;
      float3 vel;
      ///*
      float3 brnMot;
      myRandn(rseed, brnMot.y, brnMot.x, brnMot.z, u);
      if(isInside)
         vel = brnMot * intraStdDev;
      else
         vel = brnMot * extraStdDev;
      ///*
      /*
      if(isInside)
         myRandDir(rseed, intraStdDev, vel);
      else
         myRandDir(rseed, extraStdDev, vel);
      u = myRandf(rseed);
      */

      pos += vel;
      //--->num of ops untill now in the kernel = 40

      // don't let the spin leave the volume
      if (pos.x > 1.0f)  { pos.x = 1.0f; }
      else if (pos.x < -1.0f) { pos.x = -1.0f; }
      if (pos.y > 1.0f)  { pos.y = 1.0f; }
      else if (pos.y < -1.0f) { pos.y = -1.0f; }
      if (pos.z > 1.0f)  { pos.z = 1.0f; }
      else if (pos.z < -1.0f) { pos.z = -1.0f; }
   
      if(permeability<1.0f){
         reflected = false;
         float curDistToCenter;
         float3 posVec;
         float3 bounceVec = {-1.0f, -1.0f, -1.0f};

         // TO DO:
         // take account of innerRadiusScale (myelin sheath). 
         // Maybe compute the vector representing the spin position relative to the 
         // 
         if(isInside){
            float4 fiberTmp = tex1Dfetch(texFiberList, insideFiberIndex);
            float3 fiberPos = make_float3(fiberTmp);
            float fiberRadSq = fiberTmp.w;
            
            // fiberPos.x|y|z == 2 means this is the longitudial axis of the fiber
            if(fiberPos.x==2.0f)     { fiberPos.x = pos.x; bounceVec.x = 0.0f; }
            else if(fiberPos.y==2.0f){ fiberPos.y = pos.y; bounceVec.y = 0.0f; }
            else if(fiberPos.z==2.0f){ fiberPos.z = pos.z; bounceVec.z = 0.0f; }
            posVec = fiberPos-pos;
            curDistToCenter = dot(posVec,posVec);
            if(curDistToCenter>=fiberRadSq*innerRadiusScale){
               if(u>=permeability) reflected = true;
               else insideFiberIndex = UINT_MAX;
            }//num of ops = 5 for this
         }else{
            uint cubeIndex;

            cubeIndex = calcCubeHash(calcCubePos(pos,cubeLength), numCubes);   		      

            for(uint j=0;j<cubeCounters[cubeIndex];j++){
					uint curFiberIndex = tex1Dfetch(texCubeList,cubeIndex*maxFibersPerCube+j);
               float4 fiberTmp = tex1Dfetch(texFiberList, curFiberIndex);
               float3 fiberPos = make_float3(fiberTmp);
               float fiberRadSq = fiberTmp.w;
               
               if(fiberPos.x==2.0f)     { fiberPos.x = pos.x; bounceVec.x = 0.0f; }
               else if(fiberPos.y==2.0f){ fiberPos.y = pos.y; bounceVec.y = 0.0f; }
               else if(fiberPos.z==2.0f){ fiberPos.z = pos.z; bounceVec.z = 0.0f; }
               posVec = fiberPos-pos;
               curDistToCenter = dot(posVec,posVec);

               if(curDistToCenter<=fiberRadSq){                  
                  if(u>=permeability) reflected = true;
                  else insideFiberIndex = curFiberIndex;
                  break;//can break once a fiber interaction is detected
               }
            }//num of ops = 8 * 25 [38 max] (avg case)
         }

         if(reflected){
            // TO DO: approximate a bounce here!
            pos = pos+vel*bounceVec; 
         }  
      } // end if(permeability)
      
      // calculate the local magnetic field of each spin and adjust the spin phase accordingly
      phase += (gradX * pos.x + gradY * pos.y + gradZ * pos.z) * phaseConstant; //num of ops = 7
   }
   
   // store new position and velocity
   oldPos[index] = make_float4(pos,phase);
   // store new seed values
   oldSeed[index].x = rseed[0];
   oldSeed[index].y = rseed[1];
   oldSeed[index].w = insideFiberIndex; // num of ops = 5
   //TOTAL OPS for the kernel = 40 + 200 + 6 = 246 * niter 
}



__global__ void
integrateTest(  float4* oldPos, 
            float adcStdDev,          
            int numBodies,
            float gradX, float gradY, float gradZ,
            uint iterations){
            
   int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;

   if(index>=numBodies)
   return;

   float phase = oldPos[index].w;    
   float3 pos = make_float3(oldPos[index]);    

   float3 brnMot;
   uint rseed[2];
   rseed[0] = clock() + (uint)index;
   rseed[1] = rseed[0] + 1234567;
   
   for(uint i=0; i<iterations; i++){
      float u;
      myRandn(rseed, brnMot.y, brnMot.x, brnMot.z, u);
      pos += brnMot * adcStdDev;

      // bounce off cube sides
      if (pos.x > 1.0f)  { pos.x = 1.0f; }
      if (pos.x < -1.0f) { pos.x = -1.0f; }
      if (pos.y > 1.0f)  { pos.y = 1.0f; }
      if (pos.y < -1.0f) { pos.y = -1.0f; }
      if (pos.z > 1.0f)  { pos.z = 1.0f; }
      if (pos.z < -1.0f) { pos.z = -1.0f; }
   }
   
   // store new position and velocity
   oldPos[index] = make_float4(pos,phase);
}


#endif
