#include "hip/hip_runtime.h"
/* 
 * MR Diffusion GPU Device code
 * ----------------------------
 *
 * This file contains the functions (kernels) invoked by the host and to be 
 * executed by the GPU device.
 *
 * Copyright 2008 Bob Dougherty (bobd@stanford.edu) and 
 * Shyamsundar Gopalakrishnan (gshyam@stanford.edu).
 */

#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include <helper_functions.h>
#include "spinKernel.cuh"
#include "spinKernel.cu"
#include "radixsort.cu"
#include <hip/hip_runtime_api.h>
extern "C"
{

void checkCUDA()
{
  gpuDeviceInit(0);
}

void allocateArray(void **devPtr, size_t size)
{
	checkCudaErrors(hipMalloc(devPtr,size));
}

void freeArray(void *devPtr)
{
	checkCudaErrors(hipFree(devPtr));
}

void threadSync()
{
	checkCudaErrors(hipDeviceSynchronize());
}

void copyArrayFromDevice(void* host, const void* device, unsigned int vbo, int size)
{
	if (vbo)
		checkCudaErrors(cudaGLMapBufferObject((void**)&device, vbo));
	checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
	if (vbo)
		checkCudaErrors(cudaGLUnmapBufferObject(vbo));
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
	checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void registerGLBufferObject(uint vbo)
{
	checkCudaErrors(cudaGLRegisterBufferObject(vbo));
}

void unregisterGLBufferObject(uint vbo)
{
	checkCudaErrors(cudaGLUnregisterBufferObject(vbo));
}

void bindFiberList(float* ptr, int size)
{
     hipBindTexture(0,texFiberList,ptr,size*sizeof(float4));     
}

void unbindFiberList()
{
     hipUnbindTexture(texFiberList);
}

void bindCubeList(uint* ptr, int size)
{
     hipBindTexture(0,texCubeList,ptr,size*sizeof(uint));     
}

void unbindCubeList()
{
     hipUnbindTexture(texCubeList);
}


//This is the function that issues batch of threads to the device invoked by the host processor
void integrateSystem(
			         float* pos,
                     uint* randSeed,
                     float deltaTime,
                     float *fiberPos,
                     float permeability,
                     float intraAdc,
                     float extraAdc,
                     float myelinAdc,
                     int numBodies,
                     float3 gradient,
		     float phaseConstant,
                     uint* cubeCounters,
                     uint* cubeList,
                     float cubeLength,
                     uint numCubes,
                     uint maxFibersPerCube,
                     float myelinRadius,
                     uint iterations					
                     )
{

	static bool firstCall = true; //set this to true to display the GPU device specs
	struct hipDeviceProp_t devInfo; // Get some information about the device 
	hipGetDeviceProperties( &devInfo, 0 ); 

	if( firstCall ) { 
	firstCall = false;
	// Write out some info  
	printf("CUDA Device Info:\n\n"); 
	printf("Name: %s\n", devInfo.name ); 
	printf("totalGlobalMem: %u\n",devInfo.totalGlobalMem ); 
	printf("sharedMemPerBlock: %u\n",devInfo.sharedMemPerBlock );
	printf("regsPerBlock: %u\n",devInfo.regsPerBlock ); 
	printf("warpSize: %u\n",devInfo.warpSize );
	printf("memPitch %u\n",devInfo.memPitch );
	printf("maxThreadsPerBlock: %u\n",devInfo.maxThreadsPerBlock); printf("\n\n"); 
	} 

/* TO OPTIMIZE FOR GPU:
   
   1. Run separate kernels for fibers of different orientation. This 
      will eliminate the if statements in the kernel that check for the 
      longitudinal axis. Eg., if all fibers are oriented parallel to z, 
      we will run one kernel and explictly tell it that all fibers are 
      parallel to z. A simple criss-cross patten will involve running 
      twokernels sequentially. This will double the number of times we 
      loop over spins, but I suspect that it will produce a large 
      net increase in speed due to the simplfied kernel with no 
      conditionals.

   2. Sort spins by cubelocation and try to create thread blocks 
      with homogeneous cubelocations. Then, we can load the fibers for 
      that cubeinto shared memory. E.g., see the optimized nbody 
      problem in:
http://www.pas.rochester.edu/~rge21/computing/gpucomputing/cudaoptimise.shtml
      

*/

   int numThreads = min(256, numBodies);
   //int numThreads = min(devInfo.maxThreadsPerBlock, numBodies);
	int numBlocks =  1 +  (numBodies / numThreads); 

	// To avoide extra computation in the kernel, we compute the random walk 
	// standard deviation out here and pass it in.
	// The constant should be 2.0, but this leads to a slight underestimation of the 
	// mean displacement (maybe due to PRNG bias? boundary reflections?) 
    // Try: c=randn(1000000,3).*sqrt(2.0*adc*t);fprintf('%0.6f vs. %0.6f\n',mean(sqrt(sum(c.^2,2))),sqrt(6*adc*t))
    // Given this, a constant of 2.355 would seem to produce the correct displacement
	float intraStdDev = sqrt(2.0f * intraAdc * deltaTime);
	float extraStdDev = sqrt(2.0f * extraAdc * deltaTime);
	float myelinStdDev = sqrt(2.0f * myelinAdc * deltaTime);  

	//float intraStdDev = sqrt(6.0f * intraAdc * deltaTime);
	//float extraStdDev = sqrt(6.0f * extraAdc * deltaTime); 

	// execute the kernel
	integrate<<< numBlocks, numThreads>>>((float4*)pos,
		                                  (uint4*)randSeed,
		                                  deltaTime,
		                                  permeability,
		                                  intraStdDev,
		                                  extraStdDev,
						  myelinStdDev,
		                                  numBodies,
		                                  gradient.x, gradient.y, gradient.z,
		                                  (float4*)fiberPos,
		                                  cubeCounters,
		                                  cubeList,
		                                  phaseConstant,
		                                  cubeLength,
		                                  numCubes,
		                                  maxFibersPerCube,
		                                  myelinRadius*myelinRadius,
		                                  iterations);

	// check if kernel invocation generated an error
	getLastCudaError("Kernel execution failed");
}

void integrateSystemVbo(
			         uint vboPos,
                     uint* randSeed,
                     float deltaTime,
                     float *fiberPos,
                     float permeability,
                     float intraAdc,
                     float extraAdc,
		     float myelinAdc,
                     int numBodies,
                     float3 gradient,
		     float phaseConstant,
                     uint* cubeCounters,
                     uint* cubeList,
                     float cubeLength,
                     uint numCubes,
                     uint maxFibersPerCube,
                     float myelinRadius,
                     uint iterations					
                     )
{
	float *pos;
	checkCudaErrors(cudaGLMapBufferObject((void**)&pos, vboPos));
	integrateSystem(pos,randSeed,deltaTime,fiberPos,permeability,intraAdc,extraAdc,myelinAdc,numBodies,gradient,
	                phaseConstant,cubeCounters,cubeList,cubeLength,numCubes,maxFibersPerCube,
	                myelinRadius,iterations);
	//now copy back the space
	checkCudaErrors(cudaGLUnmapBufferObject(vboPos));
}


}   // extern "C"
